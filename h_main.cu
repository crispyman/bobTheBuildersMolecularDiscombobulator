#include "hip/hip_runtime.h"
//
// Created by andrewiii on 5/10/22.
//

def discombob_on_cpu(float * energyGrid, int dimX, int dimY, int dimZ, float gridSpacing, float *atoms, in numAtoms){
    int i,j,n;
    int atomArrDim = numatoms * 4;
    for (k=0; k<dimZ; k++) {
        float z = gridSpacing * (float)k;
        for (j = 0; j < dimY; j++) {
            float y = gridSpacing * (float)j;
            for (i = 0; i<dimX; i++){
                float x = gridSpacing * (float)i;
                float energy = 0.0f;
                for (n = 0; n<atomArrDim; n+=4){
                    float dx = x-atoms[n];
                    float dy = y - atoms[n+1];
                    float dz = z - atoms[n+2];
                    energy += atoms[n+3]/sqrt(dx * dx + dy * dy + dz * dz);
                }
                energyGrid[dimX*dimY*z + dimX * j + i] = energy;
            }
        }
    }

}