#include "hip/hip_runtime.h"
//
// Created by andrewiii on 5/10/22.
//
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "h_main.h"

void discombob(float * energyGrid, float *atoms, int dimX, int dimY, int dimZ, float gridSpacing, int numAtoms);


int discombob_on_cpu(float * energyGrid, float *atoms, int dimX, int dimY, int dimZ, float gridSpacing, int numAtoms){

    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    CHECK(hipEventRecord(start_cpu));

    discombob(energyGrid, atoms, dimX, dimY, dimZ, gridSpacing, numAtoms);

    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

void discombob(float * energyGrid, float *atoms, int dimX, int dimY, int dimZ, float gridSpacing, int numAtoms) {
    int i,j,k,n;
    int atomArrDim = numAtoms * 4;
    for (k=0; k<dimZ; k++) {
        float z = gridSpacing * (float)k;
        for (j = 0; j < dimY; j++) {
            float y = gridSpacing * (float)j;
            for (i = 0; i < dimX; i++){
                float x = gridSpacing * (float)i;
                float energy = 0.0f;
                for (n = 0; n<atomArrDim; n+=4){
                    float dx = x - atoms[n];
                    float dy = y - atoms[n+1];
                    float dz = z - atoms[n+2];
                    energy += atoms[n+3]/sqrt(dx*dx + dy*dy + dz*dz);
                }
                energyGrid[dimX*dimY*k + dimX*j + i] = energy;
            }
        }
    }
}