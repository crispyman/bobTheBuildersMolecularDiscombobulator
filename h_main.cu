
#include <hip/hip_runtime.h>
//
// Created by andrewiii on 5/10/22.
//

void discombob_on_cpu(float * energyGrid, unsigned int dimX, unsigned int dimY, unsigned int dimZ, float gridSpacing, float *atoms, int numAtoms){
    unsigned int i,j,k,n;
    unsigned int atomArrDim = numAtoms * 4;
    for (k=0; k<dimZ; k++) {
        float z = gridSpacing * (float)k;
        for (j = 0; j < dimY; j++) {
            float y = gridSpacing * (float)j;
            for (i = 0; i<dimX; i++){
                float x = gridSpacing * (float)i;
                float energy = 0.0f;
                for (n = 0; n<atomArrDim; n+=4){
                    float dx = x - atoms[n];
                    float dy = y - atoms[n+1];
                    float dz = z - atoms[n+2];
                    energy += atoms[n+3]/sqrtl(abs((double)dx*dx + dy*dy + dz*dz));
                }
                energyGrid[dimX*dimY*k + dimX*j + i] = energy;
            }
        }
    }
}