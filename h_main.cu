#include "hip/hip_runtime.h"
//
// Created by andrewiii on 5/10/22.
//
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "molecule.h"
#include "h_main.h"

void discombob(double * energyGrid, atom *atoms, int dimX, int dimY, int dimZ, float gridSpacing, int numAtoms);


int discombob_on_cpu(double * energyGrid, atom *atoms, int dimX, int dimY, int dimZ, float gridSpacing, int numAtoms){

    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    CHECK(hipEventRecord(start_cpu));

    discombob(energyGrid, atoms, dimX, dimY, dimZ, gridSpacing, numAtoms);

    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

void discombob(double * energyGrid, atom *atoms, int dimX, int dimY, int dimZ, float gridSpacing, int numAtoms) {
    int i,j,k,n;
    for (k=0; k<dimZ; k++) {
        float z = gridSpacing * (float)k;
        for (j = 0; j < dimY; j++) {
            float y = gridSpacing * (float)j;
            for (i = 0; i < dimX; i++){
                float x = gridSpacing * (float)i;
                double energy = 0.0;
                for (n = 0; n<numAtoms; n++){
                    double dx = (double)x - atoms[n].x;
                    double dy = (double)y - atoms[n].y;
                    double dz = (double)z - atoms[n].z;
                    double charge = atoms[n].charge;
                    energy += charge/sqrt(dx*dx + dy*dy + dz*dz);
                }
                energyGrid[dimX*dimY*k + dimX*j + i] = energy;
            }
        }
    }
}
