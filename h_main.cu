#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "CHECK.h"
#include "molecule.h"
#include "h_main.h"

void discombob(float *energyGrid, const atom *atoms, const int dimX, const int dimY, const int dimZ,
               const float gridSpacing, const int numAtoms);


int discombob_on_cpu(float *energyGrid, const atom *atoms, const int dimX, const int dimY, const int dimZ,
                     const float gridSpacing, const int numAtoms) {

    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    CHECK(hipEventCreate(&start_cpu));
    CHECK(hipEventCreate(&stop_cpu));
    CHECK(hipEventRecord(start_cpu));

    discombob(energyGrid, atoms, dimX, dimY, dimZ, gridSpacing, numAtoms);

    CHECK(hipEventRecord(stop_cpu));
    CHECK(hipEventSynchronize(stop_cpu));
    CHECK(hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu));
    return cpuMsecTime;
}

void discombob(float *energyGrid, const atom *atoms, const int dimX, const int dimY, const int dimZ,
               const float gridSpacing, const int numAtoms) {
    int i, j, k, n;
    for (k = 0; k < dimZ; k++) {
        float z = gridSpacing * (float) k;
        for (j = 0; j < dimY; j++) {
            float y = gridSpacing * (float) j;
            for (i = 0; i < dimX; i++) {
                float x = gridSpacing * (float) i;
                double energy = 0.0f;
                for (n = 0; n < numAtoms; n++) {
                    float dx = x - atoms[n].x;
                    float dy = y - atoms[n].y;
                    float dz = z - atoms[n].z;
                    float charge = atoms[n].charge;
                    energy += charge / sqrt(dx * dx + dy * dy + dz * dz);
                }
                ((float*)energyGrid)[dimX * dimY * k + dimX * j + i] = energy;
            }
        }
    }
}
