#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "CHECK.h"
#include "config.h"
#include "molecule.h"
#include "d_main.h"

__global__ void d_discombulateKernel(float * energyGrid, const float *atoms, dim3 grid, float gridspacing,
                                      int numatoms);

__global__ void d_discombulateKernelConst(float * energyGrid, dim3 grid, float gridSpacing,
                                          int numAtoms);

/* 
    d_main.cu 
    Calculates an electrostatic potential grid for the molecule passed. 
    
    energyGrid: The grid that will contain the result
    atoms: An array of all atoms of the molecule and their positions. 
    numAtoms: The number of atoms in the molecule.
*/
__constant__ float constAtoms[1024];

int d_discombobulate(float * energyGrid, float *atoms, int dimX, int dimY, int dimZ, float gridSpacing,  int numAtoms, int which){
    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;

    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));
    CHECK(hipEventRecord(start_gpu));

    /*
     * TODO: Find out why hipMemcpyToSymbol is silently failing
     */

    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constAtoms), atoms, sizeof(float) * numAtoms * 4));



    int gridSize = sizeof(float) * dimX * dimY * dimZ;
    float * d_energyGrid;
    CHECK(hipMalloc((void**)&d_energyGrid, gridSize));

    float * d_atoms;
    CHECK(hipMalloc((void**)&d_atoms, numAtoms * 4 * sizeof(int)));
    CHECK(hipMemcpy(d_atoms, atoms, numAtoms * 4 * sizeof(int), hipMemcpyHostToDevice));

    dim3 grid(dimX, dimY, dimZ);

    if (which == 0) {
        dim3 blockDim(THREADSPERBLOCK, 1, 1);
        dim3 gridDim(ceil((1.0 * dimZ) / THREADSPERBLOCK), 1, 1);
        d_discombulateKernel<<<gridDim, blockDim>>>(d_energyGrid, d_atoms, grid, gridSpacing, numAtoms);
    }
    if (which == 1) {
        dim3 blockDim(THREADSPERBLOCK, 1, 1);
        dim3 gridDim(ceil((1.0 * dimZ) / THREADSPERBLOCK), 1, 1);
        d_discombulateKernelConst<<<gridDim, blockDim>>>(d_energyGrid, grid, gridSpacing, numAtoms);
    }

    CHECK(hipMemcpy(energyGrid, d_energyGrid, gridSize, hipMemcpyDeviceToHost));

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));
    return gpuMsecTime;

}

/* 
    d_discombobulateKernel
    A kernel that calculates the electrostatic potential and stores it 
    in a float array

    energyGrid: The float array associated with the molecule 
*/
__global__ void d_discombulateKernel(float * energyGrid, const float *atoms, dim3 grid, float gridSpacing,
                                     int numAtoms) {


    int i, j, n;
    if (blockDim.x * blockIdx.x + threadIdx.x < grid.z) {
        float z = gridSpacing * (threadIdx.x + blockIdx.x * blockDim.x);
        int atomArrDim = numAtoms * 4;
        for (j = 0; j < grid.y; j++) {
            float y = gridSpacing * (float) j;
            for (i = 0; i < grid.x; i++) {
                float x = gridSpacing * (float) i;
                float energy = 0.0f;
                for (n = 0; n < atomArrDim; n += 4) {
                    float dx = x - atoms[n];
                    float dy = y - atoms[n + 1];
                    float dz = z - atoms[n + 2];
                    energy += atoms[n + 3] / sqrt(dx * dx + dy * dy + dz * dz);
                }
                energyGrid[grid.x * grid.y * (blockIdx.x * blockDim.x + threadIdx.x) + grid.x * j + i] = energy;
            }
        }
    }
}

__global__ void d_discombulateKernelConst(float * energyGrid, dim3 grid, float gridSpacing,
                                     int numAtoms) {


    int i, j, n;
    if (blockDim.x * blockIdx.x + threadIdx.x < grid.z) {
        float z = gridSpacing * (threadIdx.x + blockIdx.x * blockDim.x);
        int atomArrDim = numAtoms * 4;
        for (j = 0; j < grid.y; j++) {
            float y = gridSpacing * (float) j;
            for (i = 0; i < grid.x; i++) {
                float x = gridSpacing * (float) i;
                float energy = 0.0f;
                for (n = 0; n < atomArrDim; n += 4) {
                    float dx = x - constAtoms[n];
                    float dy = y - constAtoms[n + 1];
                    float dz = z - constAtoms[n + 2];
                    energy += constAtoms[n + 3] / sqrt(dx * dx + dy * dy + dz * dz);
                }
                energyGrid[grid.x * grid.y * (blockIdx.x * blockDim.x + threadIdx.x) + grid.x * j + i] = energy;
            }
        }
    }
}