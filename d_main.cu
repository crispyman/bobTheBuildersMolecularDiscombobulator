#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <thread>
#include <vector>
#include "CHECK.h"
#include "config.h"
#include "molecule.h"
#include "d_main.h"

__global__ void d_discombulateKernelUnoptomized(float *energyGrid, const float *atoms, const dim3 grid, const float gridspacing,
                                     const int numatoms);

__global__ void d_discombulateKernelOptimized(float *energyGrid, const float *atoms, const dim3 grid, const float gridspacing,
                                                const int numatoms);

__global__ void d_discombulateKernelConst(float *energyGrid, const dim3 grid, const float gridspacing,
                                          const int numatoms);

__device__ void kernelConstHelper(float * energygrid, dim3 grid, float gridspacing, float z, int k, int numatoms);

__global__ void d_discombulateKernelScatter(float *energyGrid, dim3 grid, float gridSpacing,
                                            int numAtoms);

__global__ void d_discombulateKernelGather(float *energyGrid, dim3 grid, float gridSpacing,
                                           int numAtoms);

__global__ void d_discombulateKernelConst3D(float *energyGrid, const dim3 grid, const float gridspacing,
                                            const int numatoms);

__global__ void d_discombulateKernelConst3DMultiGPU(float *energyGrid, dim3 grid, const float gridSpacing,
                                                    const int gpuNum, const int numAtoms);

__global__ void d_discombulateKernelCoarseningCoalescing(float *energyGrid, dim3 grid, float gridSpacing,
                                            int numAtoms);

static __global__ void emptyKernel();

int get_device_by_ptr(void *ptr);


/* 
    d_main.cu 
    Calculates an electrostatic potential grid for the molecule passed. 
    
    energyGrid: The grid that will contain the result
    atoms: An array of all atoms of the molecule and their positions. 
    numAtoms: The number of atoms in the molecule.
*/
__constant__ atom constAtoms[MAXCONSTANTATOMS];

int d_discombobulate(float *energyGrid, const atom *atoms, const int dimX, const int dimY, const int dimZ, const float gridSpacing, const int numAtoms,
                     const int which) {
    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;

    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));

    emptyKernel<<<1024, 1024>>>();

    CHECK(hipEventRecord(start_gpu));

    int gridSize = sizeof(float) * dimX * dimY * dimZ;
    float *d_energyGrid;
    CHECK(hipMalloc((void **) &d_energyGrid, gridSize));
    //zeros GPU memory since we want a zeroed energy grid to start with
    CHECK(hipMemset(d_energyGrid, 0, gridSize));

    dim3 grid(dimX, dimY, dimZ);

    // Selects which kernel to launch.
    if (which == 0) {
        atom *d_atoms;         // The array of atoms for the device.
        CHECK(hipMalloc((void **) &d_atoms, numAtoms * sizeof(atom)));
        CHECK(hipMemcpy(d_atoms, atoms, numAtoms * sizeof(atom), hipMemcpyHostToDevice));

        dim3 blockDim(THREADSPERBLOCK, 1, 1);
        dim3 gridDim(ceil((1.0 * dimZ) / THREADSPERBLOCK), 1, 1);
        d_discombulateKernelUnoptomized<<<gridDim, blockDim>>>(d_energyGrid, (float *) d_atoms, grid, gridSpacing, numAtoms);

        CHECK(hipFree(d_atoms));
    }
        // Same kernel as previous, but this time using constant memory for the atoms array.
    else if (which == 1) {
        atom *d_atoms;         // The array of atoms for the device.
        CHECK(hipMalloc((void **) &d_atoms, numAtoms * sizeof(atom)));
        CHECK(hipMemcpy(d_atoms, atoms, numAtoms * sizeof(atom), hipMemcpyHostToDevice));

        dim3 blockDim(THREADSPERBLOCK, 1, 1);
        dim3 gridDim(ceil((1.0 * dimZ) / THREADSPERBLOCK), 1, 1);
        d_discombulateKernelOptimized<<<gridDim, blockDim>>>(d_energyGrid, (float *) d_atoms, grid, gridSpacing, numAtoms);

        CHECK(hipFree(d_atoms));
    }
        // Using a const kernel.
    else if (which == 2) {
        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constAtoms), atoms, sizeof(atom) * MAXCONSTANTATOMS));

        dim3 blockDim(THREADSPERBLOCK, 1, 1);
        dim3 gridDim(ceil((1.0 * dimZ) / THREADSPERBLOCK), 1, 1);
        d_discombulateKernelConst<<<gridDim, blockDim>>>(d_energyGrid, grid, gridSpacing, numAtoms);

    }
        // Using scatter Kernel
    else if (which == 3) {


        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constAtoms), atoms, sizeof(atom) * MAXCONSTANTATOMS));

       dim3 blockDim(THREADSPERBLOCK2D_X, THREADSPERBLOCK2D_Y, 1);
        dim3 gridDim(ceil((1.0 * dimX) / THREADSPERBLOCK2D_X), ceil((1.0 * dimY) / THREADSPERBLOCK2D_Y), 1);
        d_discombulateKernelScatter<<<gridDim, blockDim>>>(d_energyGrid, grid, gridSpacing, numAtoms);

    }
    else if (which == 4) {


        CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constAtoms), atoms, sizeof(atom) * MAXCONSTANTATOMS));

        dim3 blockDim(THREADSPERBLOCK3D_X, THREADSPERBLOCK3D_Y, THREADSPERBLOCK3D_Z);
        // Number of blocks in each direction (x, y, z) is the dimension of the block in that direction/THREADSPERBLOCK3D.
        dim3 gridDim(ceil(((1.0 * dimX) / THREADSPERBLOCK3D_X) / COARSEN_FACTOR), ceil((1.0 * dimY) / THREADSPERBLOCK3D_Y),
                 ceil((1.0 * dimZ)  / THREADSPERBLOCK3D_Z));
        d_discombulateKernelCoarseningCoalescing<<<gridDim, blockDim>>>(d_energyGrid, grid, gridSpacing, numAtoms);

    }
    // Copies results to host
    CHECK(hipMemcpy(energyGrid, d_energyGrid, gridSize, hipMemcpyDeviceToHost));

    CHECK(hipFree(d_energyGrid));

    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));
    return gpuMsecTime;

}


int d_discombobulate_multi_GPU(float *energyGrid, const atom *atoms, const int dimX, const int dimY, const int dimZ, const float gridSpacing,
                               const int numAtoms) {
    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;
    int device_count = 2;

    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));

    for (int j = 0; j < device_count; j++) {
        hipSetDevice(j);
        emptyKernel<<<1024, 1024>>>();
    }
    hipSetDevice(0);
    CHECK(hipEventRecord(start_gpu));

    const int gridSize = ceil(((dimX * dimY * dimZ) / (float) device_count));

    float *d_energyGrid[device_count];

    //int grid_fraction = gridSize;


    for (int j = 0; j < device_count; j++) {
        hipSetDevice(j);
        CHECK(hipMalloc((void **) &d_energyGrid[j], gridSize * sizeof(float)));
        //zeros GPU memory since we want a zeroed energy grid to start with
        CHECK(hipMemsetAsync(d_energyGrid[j], 0, gridSize * sizeof(float)));
    }

    dim3 grid(dimX, dimY, ceil(dimZ / (float) device_count));


    // Define the dimensions of the kernel
    dim3 blockDim(THREADSPERBLOCK3D_X, THREADSPERBLOCK3D_Y, THREADSPERBLOCK3D_Z);
    // Number of blocks in each direction (x, y, z) is the dimension of the block in that direction/THREADSPERBLOCK3D.
    dim3 gridDim(ceil((1.0 * dimX) / (THREADSPERBLOCK3D_X / COARSEN_FACTOR)), ceil((1.0 * dimY) / THREADSPERBLOCK3D_Y),
                 ceil((1.0 * dimZ / (float) device_count) / THREADSPERBLOCK3D_Z));
    // Break the atoms array into smaller parts to allow for larger atom lists.
    // Break the atoms array into smaller parts to allow for larger atom lists.
    int numAtomsRemaining = numAtoms;
    for (int i = 0; i < numAtoms / MAXCONSTANTATOMS; i++) {
        // Copy atoms to constant memory on the device.
        for (int j = 0; j < device_count; j++) {
            hipSetDevice(j);
            CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(constAtoms), &atoms[i * MAXCONSTANTATOMS], sizeof(atom) * MAXCONSTANTATOMS));
            d_discombulateKernelConst3DMultiGPU<<<gridDim, blockDim>>>(d_energyGrid[j], grid, gridSpacing,
                                                                       j, MAXCONSTANTATOMS);
            numAtomsRemaining -= MAXCONSTANTATOMS;
        }

    }
    for (int j = 0; j < device_count; j++) {
        hipSetDevice(j);
        if (numAtomsRemaining < MAXCONSTANTATOMS) {
            CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(constAtoms), &atoms[numAtoms - numAtomsRemaining],
                                     sizeof(atom) * numAtomsRemaining));
            d_discombulateKernelConst3DMultiGPU<<<gridDim, blockDim>>>(d_energyGrid[j], grid, gridSpacing,
                                                                       j, numAtomsRemaining);


        }
    }

    // Copies results to host
    for (int j = 0; j < device_count; j++) {
        hipSetDevice(j);
        CHECK(hipMemcpyAsync((energyGrid + gridSize * j), d_energyGrid[j], gridSize * sizeof(float),
                         hipMemcpyDeviceToHost));
        CHECK(hipFree(d_energyGrid[j]));
    }
    hipSetDevice(0);
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));
    return gpuMsecTime;

}

int d_discombobulate_multi_GPU_threaded(float *energyGrid, const atom *atoms, const int dimX, const int dimY, const int dimZ, const float gridSpacing,
                                        const int numAtoms) {
    hipEvent_t start_gpu, stop_gpu;
    float gpuMsecTime = -1;
    int device_count = 2;

    CHECK(hipEventCreate(&start_gpu));
    CHECK(hipEventCreate(&stop_gpu));
    std::vector <std::thread> threads;

    hipSetDevice(0);
    CHECK(hipEventRecord(start_gpu));

    for (int j = device_count - 1; j >= 0; j--) {
        threads.push_back(std::thread([&, j]() {
            hipSetDevice(j);


            const int gridSize = ceil(((dimX * dimY * dimZ) / (float) device_count));

            float *d_energyGrid[device_count];

            //int grid_fraction = gridSize;



            CHECK(hipMalloc((void **) &d_energyGrid[j], gridSize * sizeof(float)));
            //zeros GPU memory since we want a zeroed energy grid to start with
            CHECK(hipMemsetAsync(d_energyGrid[j], 0, gridSize * sizeof(float)));


            dim3 grid(dimX, dimY, ceil(dimZ / (float) device_count));


            // Define the dimensions of the kernel
            dim3 blockDim(THREADSPERBLOCK3D_X, THREADSPERBLOCK3D_Y, THREADSPERBLOCK3D_Z);
            // Number of blocks in each direction (x, y, z) is the dimension of the block in that direction/THREADSPERBLOCK3D.
            dim3 gridDim(ceil((1.0 * dimX) / THREADSPERBLOCK3D_X), ceil((1.0 * dimY) / THREADSPERBLOCK3D_Y),
                         ceil((1.0 * dimZ / (float) device_count) / THREADSPERBLOCK3D_Z));
            // Break the atoms array into smaller parts to allow for larger atom lists.
            // Break the atoms array into smaller parts to allow for larger atom lists.
            int numAtomsRemaining = numAtoms;
            for (int i = 0; i < numAtoms / MAXCONSTANTATOMS; i++) {
                // Copy atoms to constant memory on the device.
                for (int j = 0; j < device_count; j++) {
                    hipSetDevice(j);
                    CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(constAtoms), &atoms[i * MAXCONSTANTATOMS],
                                             sizeof(atom) * MAXCONSTANTATOMS));
                    d_discombulateKernelConst3DMultiGPU<<<gridDim, blockDim>>>(d_energyGrid[j], grid, gridSpacing,
                                                                               j, MAXCONSTANTATOMS);
                    numAtomsRemaining -= MAXCONSTANTATOMS;
                }

            }

            if (numAtomsRemaining < MAXCONSTANTATOMS) {
                CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(constAtoms), &atoms[numAtoms - numAtomsRemaining],
                                         sizeof(atom) * numAtomsRemaining));
                d_discombulateKernelConst3DMultiGPU<<<gridDim, blockDim>>>(d_energyGrid[j], grid, gridSpacing,
                                                                           j, numAtomsRemaining);


            }


            // Copies results to host
            CHECK(hipMemcpyAsync((energyGrid + gridSize * j), d_energyGrid[j], gridSize * sizeof(float),
                             hipMemcpyDeviceToHost));
            CHECK(hipFree(d_energyGrid[j]));

        }));


    }

    for (auto &thread: threads)
        thread.join();

    hipSetDevice(0);
    CHECK(hipEventRecord(stop_gpu));
    CHECK(hipEventSynchronize(stop_gpu));
    CHECK(hipEventElapsedTime(&gpuMsecTime, start_gpu, stop_gpu));
    return gpuMsecTime;
}


/* 
    d_discombobulateKernel
    A kernel that calculates the electrostatic potential and stores it 
    in a float array

    This performed about 7x better than cpu before switching to atom struct

    energyGrid: A float array of the sample points in and around the molecule
    atoms: an array of structs that have all the necessary information about each atom, see molecule.h
    grid: a dim3 struct containing the dimensions of energyGrid
    gridSpacing: the space between grid points along each axis
    numAtoms: number of atoms in atoms array
*/
__global__ void d_discombulateKernelUnoptomized(float *energyGrid, const float *atoms, dim3 grid, float gridSpacing,
                                     int numAtoms) {

    // check to ensure thread is supposed to be doing work
    if (blockDim.x * blockIdx.x + threadIdx.x < grid.z) {
        int k = (threadIdx.x + blockIdx.x * blockDim.x);
        float z = gridSpacing * (float)k;

        int atomarrdim = numAtoms * 4;
        for (int j = 0; j < (int) grid.y; j++){
            float y = gridSpacing * (float) j;
            for (int i = 0; i < (int) grid.x; i++){
                float x = gridSpacing * (float) i;
                float energy = 0.0f;
                for (int n = 0; n < atomarrdim; n+=4) {
                    float dx = x - atoms[n];
                    float dy = y - atoms[n + 1];
                    float dz = z - atoms[n + 2];
                    energy += atoms[n + 3] / sqrtf(dx * dx + dy * dy + dz * dz);
                }
                energyGrid[grid.x * grid.y * k + grid.x * j + i] = energy;
            }
        }
    }
}

/*
    d_discombobulateKernel
    A kernel that calculates the electrostatic potential and stores it
    in a float array

    This performed about 7x better than cpu before switching to atom struct

    energyGrid: A float array of the sample points in and around the molecule
    atoms: an array of structs that have all the necessary information about each atom, see molecule.h
    grid: a dim3 struct containing the dimensions of energyGrid
    gridSpacing: the space between grid points along each axis
    numAtoms: number of atoms in atoms array
*/
__global__ void d_discombulateKernelOptimized(float *energyGrid, const float *atoms, dim3 grid, float gridSpacing,
                                                int numAtoms) {

    // check to ensure thread is supposed to be doing work
    if (blockDim.x * blockIdx.x + threadIdx.x < grid.z) {
        int k = (threadIdx.x + blockIdx.x * blockDim.x);
        float z = gridSpacing * (float)k;


        int atomarrdim = numAtoms * 4;
        int grid_slice_offset = (grid.x * grid.y * k);
        for (int n = 0; n < atomarrdim; n+=4) {
            float dz = z - atoms[n + 2];
            float dz2 = dz * dz;
            float charge = atoms[n+3];
            for (int j = 0; j < (int) grid.y; j++) {
                float y = gridSpacing * (float) j;
                float dy = y - atoms[n + 1];
                float dy2 = dy*dy;
                int grid_row_offset = grid_slice_offset + grid.x*j;
                for (int i = 0; i < (int) grid.x; i++) {
                    float x = gridSpacing * (float) i;
                    float dx = x - atoms[n];

                    energyGrid[grid_row_offset + i] += charge / sqrtf(dx * dx + dy2 + dz2);
                }
            }
        }
    }
}


/*
 * d_discombobulateKernelConst
 * A kernel that calculates the electrostatic potential and stores it in a float array,
 * this version uses constant memory instead of passing in an atoms array
 *
 * energyGrid: A float array of the sample points in and around the molecule
 * grid: a dim3 struct containing the dimensions of energyGrid
 * gridSpacing: the space between grid points along each axis
 *   numAtoms: number of atoms in constAtoms
 */

__global__ void d_discombulateKernelConst(float *energyGrid, dim3 grid, float gridSpacing,
                                          int numAtoms) {

    int k = (threadIdx.x + blockIdx.x * blockDim.x);
    float z = gridSpacing * (float) k;

    int grid_slice_offset = (grid.x * grid.y * z) / (float) gridSpacing;

    for (int n = 0; n < numAtoms; n++) {
        float dz = z - constAtoms[n].z;
        float dz2 = dz * dz;
        float charge = constAtoms[n].charge;
        for (int j = 0; j < grid.y; j++) {
            float y = gridSpacing * (float) j;
            float dy = y - constAtoms[n].y;
            float dy2 = dy * dy;
            int grid_row_offset = grid_slice_offset + grid.x * j;
            for (int i = 0; i < grid.x; i++) {
                float x = gridSpacing * (float) i;
                float dx = x - constAtoms[n].x;
                energyGrid[grid_row_offset + i] += charge / sqrtf(dx * dx + dy2 + dz2);
            }
        }
    }
}


__global__ void d_discombulateKernelScatter(float *energyGrid, dim3 grid, float gridSpacing,
                                          int numAtoms) {

    int k = (threadIdx.x + blockIdx.x * blockDim.x);
    float z = gridSpacing * (float) k;

    int n = (blockIdx.x * blockDim.x + threadIdx.x);
    float dz = z - constAtoms[n].z;
    float dz2 = dz * dz;
    int grid_slice_offset = (grid.x * grid.y * z) / gridSpacing;
    float charge = constAtoms[n].charge;
    for (int j = 0; j < grid.y; j++) {
        float y = gridSpacing * (float) j;
        float dy = y - constAtoms[n].y;
        float dy2 = dy * dy;
        int grid_row_offset = grid_slice_offset + grid.x * j;
        for (int i = 0; i < grid.x; i++) {
            float x = gridSpacing * (float) i;
            float dx = x - constAtoms[n].x;
            atomicAdd(&energyGrid[grid_row_offset + i], charge / sqrtf(dx * dx + dy2 + dz2));
        }
    }


}


__global__ void d_discombulateKernelGather(float *energyGrid, dim3 grid, float gridSpacing,
                                            int numAtoms) {

    if (blockDim.x * blockIdx.x + threadIdx.x < grid.z) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int k = (threadIdx.x + blockIdx.x * blockDim.x);
        float z = gridSpacing * (float) k;

        float y = gridSpacing * (float) j;
        float x = gridSpacing * (float) i;

        float energy = 0.0f;

        for (int n=0; n<numAtoms; n++){
            float dx = x - constAtoms[n].x;
            float dy = y - constAtoms[n].y;
            float dz = z - constAtoms[n].z;

            energy += constAtoms[n].charge / sqrtf(dx * dx + dy*dy + dz*dz);
        }

        energyGrid[grid.x * grid.y * k + grid.x * j + i] += energy;

    }
}

__global__ void d_discombulateKernelCoarseningCoalescing(float *energyGrid, dim3 grid, float gridSpacing,
                                            int numAtoms) {

    int i = blockIdx.x * blockDim.x * COARSEN_FACTOR + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = threadIdx.z + blockIdx.z * blockDim.z;
    float z = gridSpacing * (float) k;
    float y = gridSpacing * (float) j;
    float x = gridSpacing * (float) i;

    float energy0 = 0.0f;
    float energy1 = 0.0f;
    float energy2 = 0.0f;
    float energy3 = 0.0f;

    for (int n = 0; n < numAtoms; n++) {
        float dx0 = x - constAtoms[n].x;
        float dx1 = dx0 + blockDim.x * gridSpacing;
        float dx2 = dx0 + 2 *  blockDim.x * gridSpacing;
        float dx3 = dx0 + 3 *  blockDim.x * gridSpacing;
        float dy = y - constAtoms[n].y;
        float dz = z - constAtoms[n].z;
        float dysqdzsq = dy*dy + dz*dz;
        float charge = constAtoms[n].charge;
        energy0 += charge / sqrtf(dx0*dx0 + dysqdzsq);
        energy1 += charge / sqrtf(dx1*dx1 + dysqdzsq);
        energy2 += charge / sqrtf(dx2*dx2 + dysqdzsq);
        energy3 += charge / sqrtf(dx3*dx3 + dysqdzsq);
    }

    energyGrid[grid.x*grid.y*k + grid.x*j + i] += energy0;
    energyGrid[grid.x*grid.y*k + grid.x*j + i + blockDim.x] += energy1;
    energyGrid[grid.x*grid.y*k + grid.x*j + i + 2 * blockDim.x] += energy2;
    energyGrid[grid.x*grid.y*k + grid.x*j + i + 3 * blockDim.x] += energy3;

}

__device__ void kernelConstHelper(float * energygrid, dim3 grid, float gridspacing, float z, int k, int numatoms){

    int i,j,n;
    for (j = 0; j < (int)grid.y; j++){
        for (i = 0; i < (int)grid.x; i++){
            int outaddr = grid.x * grid.y * k + grid.x * j + i;
            float x = gridspacing * (float)i;
            float y = gridspacing * (float)j;
            float energy = energygrid[outaddr];
            for (n = 0; n < numatoms; n++) {
                float dx = x - constAtoms[n].x;
                float dy = y - constAtoms[n].y;
                float dz = z - constAtoms[n].z;
                energy += constAtoms[n].charge * rsqrtf(dx * dx + dy * dy + dz * dz);
            }
            energygrid[outaddr] = energy;
        }
    }
}


/*
 * d_discombobulateKernelConst2D
 * A kernel that calculates the electrostatic potential and stores it in a float array,
    this version uses constant memory instead of passing in an atoms array, Parallelizes
    on the x and y Axis
 *
 * energyGrid: A float array of the sample points in and around the molecule
 * grid: a dim3 struct containing the dimensions of energyGrid
 * gridSpacing: the space between grid points along each axis
 *   numAtoms: number of atoms in constAtoms
 */

__global__ void d_discombulateKernelConst2D(float *energyGrid, dim3 grid, float gridSpacing,
                                            int numAtoms) {

/* 
    OVERALL SUMMARY: 
    Basically the same kernel as the previous constant kernel, but in this case, each thread handles
    only one (x,y) and every z associated with that (x,y). 
*/

    int i, n;                                                   // Iterator
    int idX = blockDim.x * blockIdx.x + threadIdx.x;            // Thread x index 
    int idY = blockDim.y * blockIdx.y + threadIdx.y;            // Thread y index
    if (idX < grid.x &&
        idY < grid.y) {                         // If thread index is on the grid in both x and y direcion
        float x = gridSpacing * (float) idX;                    // X-index on the grid for the current thread.
        float y = gridSpacing * (float) idY;                    // Y-index on the grid for the current thread.
        // For each z-index into the grid
        for (i = 0; i < grid.z; i++) {                          // The z-index of the current slice. 
            // Calculate the grid index
            int gridIndex = grid.x * grid.y * i + grid.x * idY + idX;
            float z = gridSpacing * (float) i;
            float energy = 0.0f;
            // load early to offset loading time before use
            float oldEnergy = energyGrid[gridIndex];            //
            for (n = 0; n < numAtoms; n++) {
                float dx = x - constAtoms[n].x;
                float dy = y - constAtoms[n].y;
                float dz = z - constAtoms[n].z;
                float charge = constAtoms[n].charge;
                energy += charge * rsqrtf(dx * dx + dy * dy + dz * dz);

            }
            // add old and new energy values and store them
            energyGrid[gridIndex] = energy + oldEnergy;
//            __syncthreads();
        }
    }
}



__global__ void d_discombulateKernelConst3D(float *energyGrid, dim3 grid, float gridSpacing,
                                            int numAtoms) {

/* 
    OVERALL SUMMARY: 
    Basically the same kernel as the previous constant kernel, but in this case, each thread handles
    only one (x,y,z).
*/

    int n;
    // computes indexes in x, y, and z axis from block and thread
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
    int idZ = blockDim.z * blockIdx.z + threadIdx.z;

    // check to ensure thread is supposed to be doing work
    if (idX < grid.x && idY < grid.y && idZ < grid.z) {
        float x = gridSpacing * (float) idX;
        float y = gridSpacing * (float) idY;
        float z = gridSpacing * (float) idZ;
        int gridIndex = grid.x * grid.y * idZ + grid.x * idY + idX;
        float energy = 0.0f;
        // load early to offset loading time before use
        float oldEnergy = energyGrid[gridIndex];
        for (n = 0; n < numAtoms; n++) {
            float dx = x - constAtoms[n].x;
            float dy = y - constAtoms[n].y;
            float dz = z - constAtoms[n].z;
            float charge = constAtoms[n].charge;
            energy += charge * rsqrtf(dx * dx + dy * dy + dz * dz);

        }
        // add old and new energy values and store them
        energyGrid[gridIndex] += energy + oldEnergy;
    }
}

__global__ void d_discombulateKernelConst3DMultiGPU(float *energyGrid, dim3 grid, const float gridSpacing, const int gpuNum,
                                                    const int numAtoms) {

/*
    OVERALL SUMMARY:
    Basically the same kernel as the previous constant kernel, but in this case, each thread handles
    only one (x,y,z).
*/
    /*
    OVERALL SUMMARY:
    Basically the same kernel as the previous constant kernel, but in this case, each thread handles
    only one (x,y,z).
*/
    int n;
    // computes indexes in x, y, and z axis from block and thread
    int idX = blockDim.x * blockIdx.x + threadIdx.x;
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
    int idZ = blockDim.z * blockIdx.z + threadIdx.z;
    // check to ensure thread is supposed to be doing work
    if (idX < grid.x && idY < grid.y && idZ < grid.z) {
        float x = gridSpacing * (float) idX;
        float y = gridSpacing * (float) idY;
        float z = gridSpacing * (float) (idZ + grid.z * gpuNum);
        int gridIndex = grid.x * grid.y * idZ + grid.x * idY + idX;
//        __syncthreads();
        float energy = 0.0f;
        // load early to offset loading time before use
        float oldEnergy = energyGrid[gridIndex];
        for (n = 0; n < numAtoms/2 * 2; n+=2) {
//            int k = (threadIdx.x + n) % numAtoms;
            float dx = x - constAtoms[n].x;
            float dy = y - constAtoms[n].y;
            float dz = z - constAtoms[n].z;
            float charge = constAtoms[n].charge;
            float dx2 = x - constAtoms[n+1].x;
            float dy2 = y - constAtoms[n+1].y;
            float dz2 = z - constAtoms[n+1].z;
            float charge2 = constAtoms[n+1].charge;

            energy += charge * rsqrtf(dx * dx + dy * dy + dz * dz)
                    + charge2 * rsqrtf(dx2 * dx2 + dy2 * dy2 + dz2 * dz2);
        }
        if (numAtoms%2){
            float dx = x - constAtoms[numAtoms-1].x;
            float dy = y - constAtoms[numAtoms-1].y;
            float dz = z - constAtoms[numAtoms-1].z;
            float charge = constAtoms[numAtoms-1].charge;
            energy += charge * rsqrtf(dx * dx + dy * dy + dz * dz);
        }
        // add old and new energy values and store them
        energyGrid[gridIndex] += energy + oldEnergy;
    }
}

// an empty kernel to improve timing?
__global__ void emptyKernel() {
}
