#include "hip/hip_runtime.h"
//
// Created by andrewiii on 5/9/22.
//

#include <string.h>
#include <typeinfo>
#include <ctype.h>
#include <assert.h>
#include <stdlib.h>
#include "molecule.h"
#include "csvparser.h"
#include "main.h"
#include "d_main.h"
#include "h_main.h"
#include "config.h"



int getMoleculeLength(CsvRow * csvRow);
atom * readMolecule(CsvParser * csvParser, int* atomCnt);
int checkGrid(float *ref, float *check, int gridLength);

void printAtoms(atom * atoms, int numAtoms);

int main(int argc, char * argv[])
{
    // Get the file name and parse it.
    char delim = ' ';
    int numAtoms = 0;

    char* file = "stripped_alinin.pqr";
    CsvParser * csvParser = CsvParser_new(file, &delim, 0);
    // Read the molecule file and write the atoms to an array of atoms.
    atom * atoms = readMolecule(csvParser, &numAtoms);
    // Print the whole list of atoms.
    // printAtoms(atoms, numAtoms);


    CsvParser_destroy(csvParser);
    // Allocate the molecule array.
    float * molecule = (float *) malloc(sizeof(float) * 4 * numAtoms);
    float maxX = 0;
    float maxY = 0;
    float maxZ = 0;

    float minX = 0;
    float minY = 0;
    float minZ = 0;

    for (int i = 0; i < numAtoms; i++){
        printf("%s, %f, %f, %f, %f\n", atoms[i].name,
                                    atoms[i].x,
                                    atoms[i].y,
                                    atoms[i].z,
                                    atoms[i].charge);
        molecule[i * 4] = atoms[i].x;
        if (atoms[i].x > maxX)
            maxX = atoms[i].x;
        else if (atoms[i].x > maxX)
            minX = atoms[i].x;

        molecule[i * 4 + 1] = atoms[i].y;
        if (atoms[i].y > maxY)
            maxY = atoms[i].y;
        else if (atoms[i].y > maxY)
            minY = atoms[i].y;

        molecule[i * 4 + 2] = atoms[i].z;
        if (atoms[i].z > maxZ)
            maxZ = atoms[i].z;
        else if (atoms[i].z > maxZ)
            minZ = atoms[i].z;

//        if (atoms[i].name[0] == 'H')
//            molecule[i * 4 + 3] = 1.0;
//        else if (atoms[i].name[0] == 'O')
//            molecule[i * 4 + 3] = -2.0;
    }

    int dimX  = (int) ((abs(maxX) + PADDING) + (int) (abs(minX) + PADDING)) * (1/GRIDSPACING);
    int dimY  = (int) ((abs(maxY) + PADDING) + (int) (abs(minY) + PADDING)) * (1/GRIDSPACING);
    int dimZ = (int) ((abs(maxZ) + PADDING) + (int) (abs(minZ) + PADDING))* (1/GRIDSPACING);

    float * energyGrid_cpu = (float *) malloc(sizeof(float) * dimX * dimY * dimZ);
    assert(energyGrid_cpu);
    printf("%d * %d * %d = %d\n",dimX, dimY, dimZ, dimX * dimY * dimZ);

    float h_time = discombob_on_cpu(energyGrid_cpu, molecule, dimX, dimY, dimZ, GRIDSPACING, numAtoms);

    printf("\nTiming\n");
    printf("------\n");
    printf("CPU: \t\t\t\t%f msec\n", h_time);

    float * energyGrid_gpu = (float *) malloc(sizeof(float) * dimX * dimY * dimZ);
    assert(energyGrid_gpu);


    float d_time = d_discombobulate(energyGrid_gpu, molecule, dimX, dimY, dimZ, GRIDSPACING, numAtoms, 0);

    checkGrid(energyGrid_cpu, energyGrid_gpu, dimX * dimY * dimZ);
    printf("GPU (0): \t\t%f msec\n", d_time);
    float speedup = h_time/d_time;
    printf("Speedup: \t\t\t%f\n", speedup);

    float * energyGrid_gpu_const = (float *) malloc(sizeof(float) * dimX * dimY * dimZ);
    assert(energyGrid_gpu);

    float d_time_const = d_discombobulate(energyGrid_gpu_const, molecule, dimX, dimY, dimZ, GRIDSPACING, numAtoms, 1);

    checkGrid(energyGrid_cpu, energyGrid_gpu_const, dimX * dimY * dimZ);
    printf("GPU (1): \t\t%f msec\n", d_time_const);
    speedup = h_time/d_time_const;
    printf("Speedup: \t\t\t%f\n", speedup);


    free(atoms);
    free(molecule);
    free(energyGrid_cpu);
    free(energyGrid_gpu);

}

/*
    getMoleculeLength
    Assigns the number of atoms in the molecule to atomCount

    csvParser: A CsvParser that is parsing the file  that describes the molecule.
    atomCount: The number of atoms in the molecule.
*/
void getMoleculeLength(CsvParser * csvParser, int * atomCount) {
    // Make a copy of the csv parser.
    int count = 0;
    char delim = csvParser->delimiter_;
    char * file = csvParser->filePath_;
    CsvParser * countParser = CsvParser_new(file, &delim, 0);

    CsvRow * row = CsvParser_getRow(countParser);
    while (strcmp(row->fields_[0], "END") != 0) {
        if (strcmp(row->fields_[0], "ATOM") == 0) {
            count++;
        }
        row = CsvParser_getRow(countParser);
    }
    *atomCount = count;
}

/*
    readMolecule
    Reads the molecule file in the parser, parses it and makes an array of atoms.

    csvParser: A parser to parse the CSV file that contains data about the molecule.
    atomCnt: A variable to return the number of atoms in the molecule.
*/
atom * readMolecule(CsvParser * csvParser, int* atomCnt) {
    // Get the number of atoms in the molecule.
    getMoleculeLength(csvParser, atomCnt);
    // Allocate an array of atoms.
    atom * atoms = (atom *) calloc(*atomCnt, sizeof(atom));
    // Get the first row of the file.
    CsvRow * csvRow = CsvParser_getRow(csvParser);
    // delete the row because we don't need it
    CsvParser_destroy_row(csvRow);

    // printf("Number of atoms: %d", *atomCnt);
    // Loop through all lines in the file until the END record.

    for (int i = 0; i < *atomCnt; i++){
        // Skip any record that is not an atom.
        csvRow = CsvParser_getRow(csvParser);
        // printf("Name CSV: %s\nx-coord: %f\ny-coord: %f\nz-coord: %f\ncharge: %f\n", csvRow->fields_[2],
        //                                                                             strtof(csvRow->fields_[5], NULL),
        //                                                                             strtof(csvRow->fields_[6], NULL),
        //                                                                             strtof(csvRow->fields_[7], NULL),
        //                                                                              strtof(csvRow->fields_[8], NULL));

        if (strcmp(*CsvParser_getFields(csvRow), "ATOM") == 0) {

            strcpy(atoms[i].name, csvRow->fields_[2]);
            atoms[i].x = strtof(csvRow->fields_[5], NULL);
            atoms[i].y = strtof(csvRow->fields_[6], NULL);
            atoms[i].z = strtof(csvRow->fields_[7], NULL);
            atoms[i].charge = strtof(csvRow->fields_[8], NULL);
            CsvParser_destroy_row(csvRow);

        }
        // printf("Name: %s\nx-coord: %f\ny-coord: %f\nz-coord: %f\ncharge: %f\n", atoms[i].name, atoms[i].x, atoms[i].y, atoms[i].z, atoms[i].charge);
    }

    return atoms;
}


/*
    Prints an atom.
*/
void printAtoms(atom * atoms, int numAtoms) {
    for ( int i = 0; i < numAtoms; i++) {
        printf("Name: %s, \n", atoms[i].name);
        printf("X: %f, \n", atoms[i].x);
        printf("Y: %f, \n", atoms[i].y);
        printf("Z: %f, \n", atoms[i].z);
        printf("Charge: %f, \n", atoms[i].charge);
    }
}


int checkGrid(float *ref, float *check, int gridLength) {
    float*correct = (float *) ref;
    float*output = (float *) check;
    for (int i = 0; i < gridLength; i++) {
        if (output[i] != correct[i]) {
            printf("Incorrect value at [%d]\n", i);
            printf("%f != %f\n", output[i], correct[i]);

            //unixError(errorMsg);
            return 1;
        }
    }

    printf("image is correct\n");
    return 0;
}