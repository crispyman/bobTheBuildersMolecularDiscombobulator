#include "hip/hip_runtime.h"
//
// Created by andrewiii on 5/9/22.
//

#include <string.h>
#include <typeinfo>
#include <ctype.h>
#include <assert.h>
#include <stdlib.h>
#include "csvparser.h"
#include "main.h"
#include "d_main.h"
#include "h_main.h"



int getMoleculeLength(CsvRow * csvRow);
atom * readMolecule(CsvParser * csvParser, int* atomCnt);
const int padding = 10;

int main(int argc, char * argv[])
{
    // Get the file name and parse it. 
    char* file = "h2o2.xyz";
    char delim = ' ';
    // Make the parser giving the correct delimeter.
    CsvParser * csvParser = CsvParser_new(file, &delim, 0);
    int numAtoms;
    atom * atoms = readMolecule(csvParser, &numAtoms);
    CsvParser_destroy(csvParser);
    float * molecule = (float *) malloc(sizeof(float) * 3 * numAtoms);
    float maxX = 0;
    float maxY = 0;
    float maxZ = 0;

    float minX = 0;
    float minY = 0;
    float minZ = 0;

    for (int i = 0; i < numAtoms; i++){
        printf("%s, %f, %f, %f\n", atoms[i].name, atoms[i].x, atoms[i].y, atoms[i].z);
        molecule[i * 3] = atoms[i].x;
        if (atoms[i].x > maxX)
            maxX = atoms[i].x;
        else if (atoms[i].x > maxX)
            minX = atoms[i].x;

        molecule[i * 3 + 1] = atoms[i].y;
        if (atoms[i].y > maxY)
            maxY = atoms[i].y;
        else if (atoms[i].y > maxY)
            minY = atoms[i].y;

        molecule[i * 3 + 2] = atoms[i].z;
        if (atoms[i].z > maxZ)
            maxZ = atoms[i].z;
        else if (atoms[i].z > maxZ)
            minZ = atoms[i].z;
    }
    int dimX  = (int) (abs(maxX) + padding) + (int) (abs(minX) + padding);
    int dimY  = (int) (abs(maxY) + padding) + (int) (abs(minY) + padding);
    int dimZ = (int) (abs(maxZ) + padding) + (int) (abs(minZ) + padding);

    float * energyGrid = (float *) malloc(sizeof(float) * dimX * dimY * dimZ);
    float gridSpacing = 0.001;

    discombob_on_cpu(energyGrid, dimX, dimY, dimZ, gridSpacing, molecule, numAtoms);



    free(atoms);
    free(molecule);
    free(energyGrid);
}


int getMoleculeLength(CsvRow * csvRow) {
    const char **rowFields = CsvParser_getFields(csvRow);
    return strtol(rowFields[0], NULL, 10);
}

atom * readMolecule(CsvParser * csvParser, int* atomCnt) {
    CsvRow *csvRow = CsvParser_getRow(csvParser);
    int numAtoms = getMoleculeLength(csvRow);
    *atomCnt = numAtoms;
    CsvParser_destroy_row(csvRow);

    csvRow = CsvParser_getRow(csvParser);
    CsvParser_destroy_row(csvRow);

    atom *atoms = (atom *) calloc(numAtoms, sizeof(atom));
    for (int j = 0; j < numAtoms; j++) {
        csvRow = CsvParser_getRow(csvParser);
        const char **rowFields = CsvParser_getFields(csvRow);
        //if (CsvParser_getNumFields(csvRow) != 4 || rowFields[0][0] < 'A' || rowFields[0][0] > 'Z') {
        //    return NULL;
        //}
        assert(CsvParser_getNumFields(csvRow) == 4);
        strcpy(atoms[j].name, rowFields[0]);
        atoms[j].x = strtof(rowFields[1], NULL);
        atoms[j].y = strtof(rowFields[2], NULL);
        atoms[j].z = strtof(rowFields[3], NULL);
        CsvParser_destroy_row(csvRow);

    }
    return atoms;

}