#include "hip/hip_runtime.h"
//
// Created by andrewiii on 5/9/22.
//

#include <string.h>
#include <typeinfo>
#include <ctype.h>
#include <assert.h>
#include <stdlib.h>
#include "molecule.h"
#include "csvparser.h"
#include "csvwriter.h"
#include "main.h"
#include "d_main.h"
#include "h_main.h"
#include "config.h"



int main(int argc, char * argv[])
{
    // Get the file name and parse it.
    char delim = ' ';
    int numAtoms = 0;
    const char* file = "stripped_alinin.pqr";
    CsvParser * csvParser = CsvParser_new(file, &delim, 0);
    // Read the molecule file and write the atoms to an array of atoms.
    atom * atoms = readMolecule(csvParser, &numAtoms);

    CsvParser_destroy(csvParser);

    // Get the maximum and minimum coordinates in all 3 directions for any atom.
    float maxX = 0;
    float maxY = 0;
    float maxZ = 0;
    float minX = 0;
    float minY = 0;
    float minZ = 0;
    for (int i = 0; i < numAtoms; i++){
        if (atoms[i].x > maxX)
            maxX = atoms[i].x;
        else if (atoms[i].x > maxX)
            minX = atoms[i].x;

        if (atoms[i].y > maxY)
            maxY = atoms[i].y;
        else if (atoms[i].y > maxY)
            minY = atoms[i].y;

        if (atoms[i].z > maxZ)
            maxZ = atoms[i].z;
        else if (atoms[i].z > maxZ)
            minZ = atoms[i].z;
    }

    int dimX  = (int) ((abs(maxX) + PADDING) + (int) (abs(minX) + PADDING)) * (1/GRIDSPACING);
    int dimY  = (int) ((abs(maxY) + PADDING) + (int) (abs(minY) + PADDING)) * (1/GRIDSPACING);
    int dimZ = (int) ((abs(maxZ) + PADDING) + (int) (abs(minZ) + PADDING))* (1/GRIDSPACING);

    // Shift the coordinates of all atoms to be positive plus padding.
    for (int i = 0; i < numAtoms; i++) {
        atoms[i].x  += (abs(minX) + PADDING);
        atoms[i].y += (abs(minY) + PADDING);
        atoms[i].z += (abs(minZ) + PADDING);
     }

    // CPU
    float * energyGrid_cpu = (float *) malloc(sizeof(float) * dimX * dimY * dimZ);
    assert(energyGrid_cpu);
    float h_time = discombob_on_cpu(energyGrid_cpu, atoms, dimX, dimY, dimZ, GRIDSPACING, numAtoms);
    writeGrid(energyGrid_cpu, dimX * dimY * dimZ, "cpu.csv");

    printf("\nTiming\n");
    printf("------\n");
    printf("CPU: \t\t\t\t%f msec\n", h_time);

    // GPU
    float * energyGrid_gpu = (float *) malloc(sizeof(float) * dimX * dimY * dimZ);
    assert(energyGrid_gpu);

    float d_time = d_discombobulate(energyGrid_gpu, atoms, dimX, dimY, dimZ, GRIDSPACING, numAtoms, 0);

    checkGrid(energyGrid_cpu, energyGrid_gpu, dimX * dimY * dimZ, "Simple Kernel");
    printf("GPU (0): \t\t%f msec\n", d_time);
    float speedup = h_time/d_time;
    printf("Speedup: \t\t\t%f\n", speedup);
    writeGrid(energyGrid_gpu, dimX * dimY * dimZ, "gpusimple.csv");


    // GPU Const
    d_time = 0;
    memset(energyGrid_gpu, 0 , sizeof(float) * dimX * dimY * dimZ);

    d_time = d_discombobulate(energyGrid_gpu, atoms, dimX, dimY, dimZ, GRIDSPACING, numAtoms, 1);

    checkGrid(energyGrid_cpu, energyGrid_gpu, dimX * dimY * dimZ, "1D Const Kernel");
    printf("GPU(1): \t\t%f msec\n", d_time);
    speedup = h_time/d_time;
    printf("Speedup: \t\t\t%f\n", speedup);
    writeGrid(energyGrid_gpu, dimX * dimY * dimZ, "gpu1D.csv");


    // GPU Const 2D
    d_time = 0;
    memset(energyGrid_gpu, 0 , sizeof(float) * dimX * dimY * dimZ);

    d_time = d_discombobulate(energyGrid_gpu, atoms, dimX, dimY, dimZ, GRIDSPACING, numAtoms, 2);

    checkGrid(energyGrid_cpu, energyGrid_gpu, dimX * dimY * dimZ, "2D Const Kernel");
    printf("GPU (2): \t\t%f msec\n", d_time);
    speedup = h_time/d_time;
    printf("Speedup: \t\t\t%f\n", speedup);
    writeGrid(energyGrid_gpu, dimX * dimY * dimZ, "gpu2D.csv");


    // GPU Const 3D
    d_time = 0;
    memset(energyGrid_gpu, 0 , sizeof(float) * dimX * dimY * dimZ);

    d_time = d_discombobulate(energyGrid_gpu, atoms, dimX, dimY, dimZ, GRIDSPACING, numAtoms, 3);

    checkGrid(energyGrid_cpu, energyGrid_gpu, dimX * dimY * dimZ, "3D Const Kernel");
    printf("GPU (3): \t\t%f msec\n", d_time);
    speedup = h_time/d_time;
    printf("Speedup: \t\t\t%f\n", speedup);
    writeGrid(energyGrid_gpu, dimX * dimY * dimZ, "gpu3D.csv");


    free(atoms); 
    //free(molecule);

    free(energyGrid_cpu);
    free(energyGrid_gpu);


}

/*
    getMoleculeLength
    Assigns the number of atoms in the molecule to atomCount

    filepath: A path to the pqr file describing the molecule

    returns: 
        count: A count of the number of 'ATOM' records in the pqr file.

*/
int getMoleculeLength(char * filepath) {
    // Open the file. 
    char str[20];
    FILE *fptr;
    fptr = fopen(filepath, "r");


    char *pos;
    int index, count;
    
    count = 0;
    // Read from the file until we reach the end.
    while ((fgets(str, 20, fptr)) != NULL)
    {
        index = 0;
        // While strstr doesnt return NULL.
        // Index and pos are needed to make so this is not an
        // infinite loop. strstr simply returns the 
        while ((pos = strstr(str + index, "ATOM")) != NULL)
        {
            // Update the current index to be the location
            // ATOM was found and increment by 1 to avoid 
            // recounting it. 
            index = (pos - str) + 1;
            count++;
        }
    }
    return count;
} 

/*
    readMolecule
    Reads the molecule file in the parser, parses it and makes an array of atoms.

    csvParser: A parser to parse the CSV file that contains data about the molecule.
    atomCnt: A variable to return the number of atoms in the molecule.
*/
atom * readMolecule(CsvParser * csvParser, int* atomCnt) {
    // Get the number of atoms in the molecule.
    *atomCnt = getMoleculeLength(csvParser->filePath_);
    // Allocate an array of atoms.
    atom * atoms = (atom *) calloc(*atomCnt, sizeof(atom));
    // Get the first row of the file.
    CsvRow * csvRow = CsvParser_getRow(csvParser);
    // delete the row because we don't need it
    CsvParser_destroy_row(csvRow);

    // Loop through all lines in the file until the END record.
    for (int i = 0; i < *atomCnt; i++){
        // Skip any record that is not an atom.
        csvRow = CsvParser_getRow(csvParser);
        if (strcmp(*CsvParser_getFields(csvRow), "ATOM") == 0) {
            atoms[i].x = strtof(csvRow->fields_[5], NULL);
            atoms[i].y = strtof(csvRow->fields_[6], NULL);
            atoms[i].z = strtof(csvRow->fields_[7], NULL);
            atoms[i].charge = strtof(csvRow->fields_[8], NULL);
            CsvParser_destroy_row(csvRow);
        }
    }
    return atoms;
}


/*
    Prints an atom.
*/
void printAtoms(atom * atoms, int numAtoms) {
    for ( int i = 0; i < numAtoms; i++) {
        //printf("Name: %s, \n", atoms[i].name);
        printf("X: %f, \n", atoms[i].x);
        printf("Y: %f, \n", atoms[i].y);
        printf("Z: %f, \n", atoms[i].z);
        printf("Charge: %f, \n", atoms[i].charge);
    }
}

/* 
    fequal: Returns 1 if the two floating point values are more different than a threshold.
*/
int fequal(float a, float b) {
    float diff = abs(a - b);
    if ((diff < PRECISIONTHRESH)  || (isinf(a) && isinf(b))) {
        // Equal
        return 0;
    }
    // Not equal.
    return 1;
}

int checkGrid(float *ref, float *check, int gridLength, const char* kernelName) {
    float*correct = (float *) ref;
    float*output = (float *) check;
    for (int i = 0; i < gridLength; i++) {
        if (fequal(correct[i], ref[i])) {
            printf("\e[1;31m%s\e[0m priduced an incorrect value at [%d]\n",kernelName, i);
            printf("Actual: %f != Expected: %f\n", output[i], correct[i]);
            return 1;
        }
    }

    printf("\e[1;32m%s\e[0m produced a correct grid.\n", kernelName);
    return 0;
}


void writeGrid(float * data, int gridLength, const char* fileName){
    char buf[1024];
    float max = 1;
    CsvWriter *csvWriter = CsvWriter_new(fileName, ",", 0);
    for (int i = 0; i < gridLength; i++){
        if (data[i] > max)
            max = data[i];
        gcvt(data[i], 25, buf);
        if (CsvWriter_writeField(csvWriter, buf)) {
            printf("Error: %s\n", CsvWriter_getErrorMessage(csvWriter));
            break;
        }
    }
    CsvWriter_destroy(csvWriter);
    // printf("\n%f\n", max);
}